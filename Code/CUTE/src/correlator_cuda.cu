#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////
//                                                                   //
//   Copyright 2012 David Alonso                                     //
//                                                                   //
//                                                                   //
// This file is part of CUTE.                                        //
//                                                                   //
// CUTE is free software: you can redistribute it and/or modify it   //
// under the terms of the GNU General Public License as published by //
// the Free Software Foundation, either version 3 of the License, or //
// (at your option) any later version.                               //
//                                                                   //
// CUTE is distributed in the hope that it will be useful, but       //
// WITHOUT ANY WARRANTY; without even the implied warranty of        //
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU //
// General Public License for more details.                          //
//                                                                   //
// You should have received a copy of the GNU General Public License //
// along with CUTE.  If not, see <http://www.gnu.org/licenses/>.     //
//                                                                   //
///////////////////////////////////////////////////////////////////////

/*********************************************************************/
//                      Correlators with CUDA-C                      //
/*********************************************************************/
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sm_20_atomic_functions.h>
#include "define.h"
#include "correlator_cuda.h"

int n_blocks;
__constant__ int cst_nside_x;
__constant__ int cst_nside_y;
__constant__ int cst_nside_z;
__constant__ int cst_irange_x;
__constant__ int cst_irange_y;
__constant__ int cst_irange_z;
__constant__ float cst_l_box_x;
__constant__ float cst_l_box_y;
__constant__ float cst_l_box_z;
__constant__ float cst_x_min;
__constant__ float cst_y_min;
__constant__ float cst_z_min;

__constant__ int cst_nside_cth;
__constant__ int cst_nside_phi;
__constant__ float cst_cth_min;
__constant__ float cst_cth_max;
__constant__ float cst_thmax;

__device__ void get_bounds(float *pos,int *bounds)
{
  float r,cth,phi;
  int icth,iphi;
  float theta,th_hi,th_lo;
  float phi_hi,phi_lo;
  float cth_max,cth_min;
  r=sqrtf(pos[0]*pos[0]+pos[1]*pos[1]+pos[2]*pos[2]);

  if(r==0) {
    cth=1;
    phi=0;
  }
  else {
    float xn,yn;
    xn=pos[0]/r;
    yn=pos[1]/r;
    cth=pos[2]/r;
    
    if((xn==0)&&(yn==0))
      phi=0;
    else {
      phi=atan2(yn,xn);
      if(phi<0) phi=2*M_PI+phi;
    }
  }
  
  if(cth>=1)
    icth=cst_nside_cth-1;
  else
    icth=(int)(0.5*(1+cth)*cst_nside_cth);

  iphi=(int)(0.5*phi/M_PI*cst_nside_phi);

  theta=acosf(-1.0+2.0*((float)(icth+0.5))/cst_nside_cth);
  th_hi=acosf(-1.0+2.0*((float)(icth+0.0))/cst_nside_cth);
  th_lo=acosf(-1.0+2.0*((float)(icth+1.0))/cst_nside_cth);
  phi_hi=2*M_PI*((float)(iphi+1.0)/cst_nside_phi);
  phi_lo=2*M_PI*((float)(iphi+0.0)/cst_nside_phi);

  if(th_hi>M_PI-cst_thmax) {
    cth_min=-1;
    cth_max=cosf(th_lo-cst_thmax);

    bounds[2]=0;
    bounds[3]=cst_nside_phi-1;
  }
  else if(th_lo<cst_thmax) {
    cth_min=cosf(th_hi+cst_thmax);
    cth_max=1;

    bounds[2]=0;
    bounds[3]=cst_nside_phi-1;
  }
  else {
    float dphi;
    float calpha=cosf(cst_thmax);
    cth_min=cosf(th_hi+cst_thmax);
    cth_max=cosf(th_lo-cst_thmax);

    if(theta<0.5*M_PI) {
      float c_thlo=cosf(th_lo);
      dphi=acosf(sqrtf((calpha*calpha-c_thlo*c_thlo)/
		       (1-c_thlo*c_thlo)));
    }
    else {
      float c_thhi=cosf(th_hi);
      dphi=acosf(sqrtf((calpha*calpha-c_thhi*c_thhi)/
		       (1-c_thhi*c_thhi)));
    }

    if(dphi<M_PI) {
      float phi_max,phi_min;
      phi_min=phi_lo-dphi;
      phi_max=phi_hi+dphi;
      bounds[2]=(int)(0.5*phi_min/M_PI*cst_nside_phi);
      bounds[3]=(int)(0.5*phi_max/M_PI*cst_nside_phi);
    }
    else {
      bounds[2]=0;
      bounds[3]=cst_nside_phi-1;
    }
  }

  //Cut with mask
  cth_min=MAX((cth_min),(cst_cth_min));
  cth_max=MIN((cth_max),(cst_cth_max));

  bounds[0]=(int)(0.5*(1+cth_min)*cst_nside_cth);
  bounds[1]=(int)(0.5*(1+cth_max)*cst_nside_cth);
  if(bounds[0]<0) bounds[0]=0;
  if(bounds[1]>=cst_nside_cth) bounds[1]=cst_nside_cth-1;
}

__global__ void cudaCrossAng(int np,float *box_pos1,
			     int *box_np2,int *box_ind2,float *box_pos2,
			     unsigned long long *hh)
{
  //////
  // Cross-correlator for angular correlation function
  // (brute-force)
  __shared__ unsigned long long hthread[NB_HISTO_1D];
  int ii;
  int stride=blockDim.x*gridDim.x;
  
  // Initialize shared histogram
  hthread[threadIdx.x]=0;
  __syncthreads();
  // Correlate

  ii=threadIdx.x+blockIdx.x*blockDim.x;
  while(ii<np) {
    float *pos1=&(box_pos1[3*ii]);
    int bounds[4];
    int icth;

    get_bounds(pos1,bounds);
    for(icth=bounds[0];icth<=bounds[1];icth++) {
      int icth_n=icth*cst_nside_phi;
      int iphi;
      for(iphi=bounds[2];iphi<=bounds[3];iphi++) {
	int jj;
	int iphi_true=(iphi+cst_nside_phi)%cst_nside_phi;
	int ibox=iphi_true+icth_n;
	int np2=box_np2[ibox];
	float *pos2=&(box_pos2[3*box_ind2[ibox]]);
	for(jj=0;jj<np2;jj++) {
	  int ibin;
	  float prod=pos1[0]*pos2[3*jj]+
	    pos1[1]*pos2[3*jj+1]+pos1[2]*pos2[3*jj+2];
#ifdef _LOGBIN
	  if(prod<1) {
#ifdef _TRUE_ACOS
	    prod=log10(acosf(prod));
#else //_TRUE_ACOS
	    prod=1-prod;
	    prod=2*prod+0.33333333333*prod*prod+
	      0.088888888889*prod*prod*prod;
	    prod=0.5*log10(prod);
#endif //_TRUE_ACOS
	    ibin=(int)(N_LOGINT*(prod-LOG_TH_MAX)+NB_HISTO_1D);
	  }
#else //_LOGBIN
#ifdef _TRUE_ACOS
	  prod=acosf((MIN(1,prod)));
#else
	  prod=1-MIN(1,prod);
	  prod=sqrtf(2*prod+0.333333333*prod*prod+
		     0.0888889*prod*prod*prod);
#endif //_TRUE_ACOS
	  ibin=(int)(prod*I_THETA_MAX*NB_HISTO_1D);
#endif //_LOGBIN
	  if((ibin<NB_HISTO_1D)&&(ibin>=0))
	    atomicAdd(&(hthread[ibin]),1);
	}
      }
    }
    ii+=stride;
  }
  // Add block histograms
  __syncthreads();
  atomicAdd(&(hh[threadIdx.x]),hthread[threadIdx.x]);
}

__global__ void cudaCrossAngPM(int npx,int *pix_full,
			       float *pos,int *npD,int *npR,
			       unsigned long long *DD,
			       unsigned long long *DR,
			       unsigned long long *RR)
{
  //////
  // Cross-correlator for angular correlation function
  // with pixelization
  __shared__ unsigned long long DDthread[NB_HISTO_1D];
  __shared__ unsigned long long DRthread[NB_HISTO_1D];
  __shared__ unsigned long long RRthread[NB_HISTO_1D];
  int ii;
  int stride=blockDim.x*gridDim.x;

  // Initialize shared histogram
  DDthread[threadIdx.x]=0;
  DRthread[threadIdx.x]=0;
  RRthread[threadIdx.x]=0;
  __syncthreads();
  // Correlate
  ii=threadIdx.x+blockIdx.x*blockDim.x;
  while(ii<npx) {
    float *pos1=&(pos[3*ii]);
    int bounds[4];
    int icth;

    get_bounds(pos1,bounds);
    for(icth=bounds[0];icth<=bounds[1];icth++) {
      int iphi;
      int icth_n=icth*cst_nside_phi;
      for(iphi=bounds[2];iphi<=bounds[3];iphi++) {
	int iphi_true=(iphi+cst_nside_phi)%cst_nside_phi;
	int ipix=iphi_true+icth_n;
	int jj=pix_full[ipix];
	if(jj!=-1) {
	  int ibin;
	  float prod=pos1[0]*pos[3*jj]+
	    pos1[1]*pos[3*jj+1]+pos1[2]*pos[3*jj+2];
	
#ifdef _LOGBIN
	  if(prod<1) {
#ifdef _TRUE_ACOS
	    prod=log10(acosf(prod));
#else //_TRUE_ACOS
	    prod=1-prod;
	    prod=2*prod+0.33333333333*prod*prod+
	      0.088888888889*prod*prod*prod;
	    prod=0.5*log10(prod);
#endif //_TRUE_ACOS
	    ibin=(int)(N_LOGINT*(prod-LOG_TH_MAX)+NB_HISTO_1D);
	  }
#else //_LOGBIN
#ifdef _TRUE_ACOS
	  prod=acosf((MIN(1,prod)));
#else
	  prod=1-MIN(1,prod);
	  prod=sqrtf(2*prod+0.333333333*prod*prod+
		     0.0888889*prod*prod*prod);
#endif //_TRUE_ACOS
	  ibin=(int)(prod*I_THETA_MAX*NB_HISTO_1D);
#endif //_LOGBIN
	  if((ibin<NB_HISTO_1D)&&(ibin>=0)) {
	    int ndd=npD[ii]*npD[jj];
	    int ndr=npD[ii]*npR[jj];
	    int nrr=npR[ii]*npR[jj];
	    atomicAdd(&(DDthread[ibin]),ndd);
	    atomicAdd(&(DRthread[ibin]),ndr);
	    atomicAdd(&(RRthread[ibin]),nrr);
	  }
	}
      }
    }
    ii+=stride;
  }

  // Add block histograms
  __syncthreads();
  atomicAdd(&(DD[threadIdx.x]),DDthread[threadIdx.x]);
  atomicAdd(&(DR[threadIdx.x]),DRthread[threadIdx.x]);
  atomicAdd(&(RR[threadIdx.x]),RRthread[threadIdx.x]);
}

__global__ void cudaCrossMono(int np,float *box_pos1,
			      int *box_np2,int *box_ind2,float *box_pos2,
			      unsigned long long *hh)
{
  //////
  // Cross-correlator for monopole 2PCF (brute-force)
  __shared__ unsigned long long hthread[NB_HISTO_1D];
  int ii;
  int stride=blockDim.x*gridDim.x;
  
  // Initialize shared histogram
  hthread[threadIdx.x]=0;
  __syncthreads();
  // Correlate
  
  ii=threadIdx.x+blockIdx.x*blockDim.x;
  while(ii<np) {
    float *pos1=&(box_pos1[3*ii]);
    
    int ix1=(int)((pos1[0]-cst_x_min)/cst_l_box_x*cst_nside_x);
    int iy1=(int)((pos1[1]-cst_y_min)/cst_l_box_y*cst_nside_y);
    int iz1=(int)((pos1[2]-cst_z_min)/cst_l_box_z*cst_nside_z);

    int ixmin=MAX(ix1-cst_irange_x,0);
    int ixmax=MIN(ix1+cst_irange_x,cst_nside_x-1);
    int iymin=MAX(iy1-cst_irange_y,0);
    int iymax=MIN(iy1+cst_irange_y,cst_nside_y-1);
    int izmin=MAX(iz1-cst_irange_z,0);
    int izmax=MIN(iz1+cst_irange_z,cst_nside_z-1);

    int iz;
    for(iz=izmin;iz<=izmax;iz++) {
      int iy;
      int iz_n=iz*cst_nside_x*cst_nside_y;
      for(iy=iymin;iy<=iymax;iy++) {
	int ix;
	int iy_n=iy*cst_nside_x;
	for(ix=ixmin;ix<=ixmax;ix++) {
	  int i2;
	  int ip2=ix+iy_n+iz_n;
	  int np2=box_np2[ip2];
	  float *pos2=&(box_pos2[3*box_ind2[ip2]]);
	  for(i2=0;i2<np2;i2++) {
	    int ibin;
	    float xd[3],r2;
	    xd[0]=pos1[0]-pos2[3*i2];
	    xd[1]=pos1[1]-pos2[3*i2+1];
	    xd[2]=pos1[2]-pos2[3*i2+2];
	    r2=xd[0]*xd[0]+xd[1]*xd[1]+xd[2]*xd[2];
#ifdef _LOGBIN
	    if(r2>0)
	      ibin=(int)(N_LOGINT*(0.5*log10(r2)-LOG_R_MAX)+NB_HISTO_1D);
	    else
	      ibin=-1;
#else //_LOGBIN
	    ibin=(int)(sqrtf(r2)*I_R_MAX*NB_HISTO_1D);
#endif //_LOGBIN
	    if((ibin<NB_HISTO_1D)&&(ibin>=0))
	      atomicAdd(&(hthread[ibin]),1);
	  }
	}
      }
    }
    ii+=stride;
  }

  // Add block histograms
  __syncthreads();
  atomicAdd(&(hh[threadIdx.x]),hthread[threadIdx.x]);
}

__global__ void cudaCross3Dps(int np,float *box_pos1,
			      int *box_np2,int *box_ind2,float *box_pos2,
			      unsigned long long *hh,int iter)
{
  //////
  // Cross-correlator for anisotropic 3-D correlation function
  // (binning in pi-sigma)
  __shared__ unsigned long long hthread[NB_X_BATCH][NB_HISTO_2D];
  __shared__ float rt20,rt2f;
  __shared__ int irt_off;
  int ii;
  int stride=blockDim.x*blockDim.y*gridDim.x;
  
  // Initialize shared histogram
  for(ii=0;ii<NB_X_BATCH/NTH_RWS_2D;ii++)
    hthread[ii*NTH_RWS_2D+threadIdx.y][threadIdx.x]=0;
  if((threadIdx.y==0)&&(threadIdx.x==0)) {
    irt_off=iter*NB_X_BATCH; //This is the first unfilled bin
                              //of the full histogram
    rt20=irt_off/(I_RT_MAX*NB_HISTO_2D); 
    rt2f=(irt_off+NB_X_BATCH)/(I_RT_MAX*NB_HISTO_2D);
    rt20=rt20*rt20; //The first unprobed transverse scale (squared)
    rt2f=rt2f*rt2f; //The last transverse scale to be probed 
                    //in this iteration
  }
  __syncthreads();

  // Correlate
  ii=threadIdx.x+threadIdx.y*blockDim.x+
    blockIdx.x*blockDim.x*blockDim.y;
  while(ii<np) {
    float *pos1=&(box_pos1[3*ii]);
    
    int ix1=(int)((pos1[0]-cst_x_min)/cst_l_box_x*cst_nside_x);
    int iy1=(int)((pos1[1]-cst_y_min)/cst_l_box_y*cst_nside_y);
    int iz1=(int)((pos1[2]-cst_z_min)/cst_l_box_z*cst_nside_z);

    int ixmin=MAX(ix1-cst_irange_x,0);
    int ixmax=MIN(ix1+cst_irange_x,cst_nside_x-1);
    int iymin=MAX(iy1-cst_irange_y,0);
    int iymax=MIN(iy1+cst_irange_y,cst_nside_y-1);
    int izmin=MAX(iz1-cst_irange_z,0);
    int izmax=MIN(iz1+cst_irange_z,cst_nside_z-1);

    int iz;
    for(iz=izmin;iz<=izmax;iz++) {
      int iy;
      int iz_n=iz*cst_nside_x*cst_nside_y;
      for(iy=iymin;iy<=iymax;iy++) {
	int ix;
	int iy_n=iy*cst_nside_x;
	for(ix=ixmin;ix<=ixmax;ix++) {
	  int i2;
	  int ip2=ix+iy_n+iz_n;
	  int np2=box_np2[ip2];
	  float *pos2=&(box_pos2[3*box_ind2[ip2]]);
	  for(i2=0;i2<np2;i2++) {
	    int irl,irt;
	    float xr[3],xcm[3];
	    float r2,rl,rt2;
	    xr[0]=pos1[0]-pos2[3*i2];
	    xr[1]=pos1[1]-pos2[3*i2+1];
	    xr[2]=pos1[2]-pos2[3*i2+2];
	    xcm[0]=0.5*(pos1[0]+pos2[3*i2]);
	    xcm[1]=0.5*(pos1[1]+pos2[3*i2+1]);
	    xcm[2]=0.5*(pos1[2]+pos2[3*i2+2]);
	    rl=fabs(xr[0]*xcm[0]+xr[1]*xcm[1]+xr[2]*xcm[2])*
	      rsqrtf(xcm[0]*xcm[0]+xcm[1]*xcm[1]+xcm[2]*xcm[2]);
	    irl=(int)(rl*I_RL_MAX*NB_HISTO_2D);
	    if(irl<NB_HISTO_2D) {
	      r2=xr[0]*xr[0]+xr[1]*xr[1]+xr[2]*xr[2];
	      rt2=r2-rl*rl;
	      if((rt2<rt2f)&&(rt2>=rt20)) {
		irt=(int)(sqrtf(rt2)*I_RT_MAX*NB_HISTO_2D)-irt_off;
		if((irt>=0)&&(irt<NB_X_BATCH))
		  atomicAdd(&(hthread[irt][irl]),1);
	      }
	    }
	  }
	}
      }
    }
    ii+=stride;
  }

  // Add block histograms
  __syncthreads();
  for(ii=0;ii<NB_X_BATCH/NTH_RWS_2D;ii++) {
    atomicAdd(&(hh[threadIdx.x+(irt_off+ii*NTH_RWS_2D+threadIdx.y)*NB_HISTO_2D]),
		hthread[ii*NTH_RWS_2D+threadIdx.y][threadIdx.x]);
  }
}

__global__ void cudaCross3Drm(int np,float *box_pos1,
			      int *box_np2,int *box_ind2,float *box_pos2,
			      unsigned long long *hh,int iter)
{
  //////
  // Cross-correlator for anisotropic 3-D correlation function
  // (binning in r-mu)
  __shared__ unsigned long long hthread[NB_X_BATCH][NB_HISTO_2D];
  __shared__ float cth0,cthf;
  __shared__ int irt_off;
  int ii;
  int stride=blockDim.x*blockDim.y*gridDim.x;
  
  // Initialize shared histogram
  for(ii=0;ii<NB_X_BATCH/NTH_RWS_2D;ii++)
    hthread[ii*NTH_RWS_2D+threadIdx.y][threadIdx.x]=0;
  if((threadIdx.x==0)&&(threadIdx.y==0)) {
    irt_off=iter*NB_X_BATCH;
    cth0=(float)irt_off/NB_HISTO_2D;
    cthf=(float)(irt_off+NB_X_BATCH)/NB_HISTO_2D;
  }
  __syncthreads();

  // Correlate
  ii=threadIdx.x+threadIdx.y*blockDim.x+
    blockIdx.x*blockDim.x*blockDim.y;
  while(ii<np) {
    float *pos1=&(box_pos1[3*ii]);
    
    int ix1=(int)((pos1[0]-cst_x_min)/cst_l_box_x*cst_nside_x);
    int iy1=(int)((pos1[1]-cst_y_min)/cst_l_box_y*cst_nside_y);
    int iz1=(int)((pos1[2]-cst_z_min)/cst_l_box_z*cst_nside_z);

    int ixmin=MAX(ix1-cst_irange_x,0);
    int ixmax=MIN(ix1+cst_irange_x,cst_nside_x-1);
    int iymin=MAX(iy1-cst_irange_y,0);
    int iymax=MIN(iy1+cst_irange_y,cst_nside_y-1);
    int izmin=MAX(iz1-cst_irange_z,0);
    int izmax=MIN(iz1+cst_irange_z,cst_nside_z-1);

    int iz;
    for(iz=izmin;iz<=izmax;iz++) {
      int iy;
      int iz_n=iz*cst_nside_x*cst_nside_y;
      for(iy=iymin;iy<=iymax;iy++) {
	int ix;
	int iy_n=iy*cst_nside_x;
	for(ix=ixmin;ix<=ixmax;ix++) {
	  int i2;
	  int ip2=ix+iy_n+iz_n;
	  int np2=box_np2[ip2];
	  float *pos2=&(box_pos2[3*box_ind2[ip2]]);
	  for(i2=0;i2<np2;i2++) {
	    int ir,icth;
	    float xr[3],xcm[3];
	    float r2,cth;
	    xr[0]=pos1[0]-pos2[3*i2];
	    xr[1]=pos1[1]-pos2[3*i2+1];
	    xr[2]=pos1[2]-pos2[3*i2+2];
	    xcm[0]=0.5*(pos1[0]+pos2[3*i2]);
	    xcm[1]=0.5*(pos1[1]+pos2[3*i2+1]);
	    xcm[2]=0.5*(pos1[2]+pos2[3*i2+2]);
	    r2=xr[0]*xr[0]+xr[1]*xr[1]+xr[2]*xr[2];
#ifdef _LOGBIN
	    if(r2==0) ir=-1;
	    else
	      ir=(int)(N_LOGINT*(0.5*log10(r2)-LOG_R3D_MAX)+NB_HISTO_2D);
#else //_LOGBIN
	    ir=(int)(sqrtf(r2)*I_R3D_MAX*NB_HISTO_2D);
#endif //_LOGBIN
	    if((ir>=0)&&(ir<NB_HISTO_2D)) {
	      cth=fabs(xr[0]*xcm[0]+xr[1]*xcm[1]+xr[2]*xcm[2])*
		rsqrtf(r2*(xcm[0]*xcm[0]+xcm[1]*xcm[1]+xcm[2]*xcm[2]));
	      if((cth<=cthf)&&(cth>cth0)) {
		icth=(int)(cth*NB_HISTO_2D)-irt_off;
		if((icth>=0)&&(icth<NB_X_BATCH))
		  atomicAdd(&(hthread[icth][ir]),1);
	      }
	    }
	  }
	}
      }
    }
    ii+=stride;
  }

  // Add block histograms
  __syncthreads();
  for(ii=0;ii<NB_X_BATCH/NTH_RWS_2D;ii++) {
    atomicAdd(&(hh[threadIdx.x+(irt_off+ii*NTH_RWS_2D+threadIdx.y)*NB_HISTO_2D]),
	      hthread[ii*NTH_RWS_2D+threadIdx.y][threadIdx.x]);
  }
}

void corr_CUDA_AngPM(float cth_min,float cth_max,
		     int npix,int *pix_full,
		     float *pos,int *npD,int *npR,
		     unsigned long long *DD,
		     unsigned long long *DR,
		     unsigned long long *RR)
{
  //////
  // Auto-correlator for angular 2PCF with brute-force
  float *pos_dev;
  int *npD_dev,*npR_dev,*pix_full_dev;
  unsigned long long *DD_dev;
  unsigned long long *DR_dev;
  unsigned long long *RR_dev;
  int ii;

  hipEvent_t start, stop;
  float elaptime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for(ii=0;ii<NB_HISTO_1D;ii++) {
    DD[ii]=0;
    DR[ii]=0;
    RR[ii]=0;
  }

  float thmax=1/I_THETA_MAX;
  hipMemcpyToSymbol(HIP_SYMBOL(cst_nside_cth),&(n_side_cth),sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_nside_phi),&(n_side_phi),sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_cth_min),&(cth_min),sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_cth_max),&(cth_max),sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_thmax),&(thmax),sizeof(float));

  //Allocate GPU memory and copy particle positions
  hipMalloc((void**)&pos_dev,3*npix*sizeof(float));
  hipMemcpy(pos_dev,pos,3*npix*sizeof(float),hipMemcpyHostToDevice);
  hipMalloc((void**)&npD_dev,npix*sizeof(int));
  hipMemcpy(npD_dev,npD,npix*sizeof(int),hipMemcpyHostToDevice);
  hipMalloc((void**)&npR_dev,npix*sizeof(int));
  hipMemcpy(npR_dev,npR,npix*sizeof(int),hipMemcpyHostToDevice);
  hipMalloc((void**)&pix_full_dev,n_boxes2D*sizeof(int));
  hipMemcpy(pix_full_dev,pix_full,n_boxes2D*sizeof(int),hipMemcpyHostToDevice);
  //Allocate GPU memory for the GPU histogram
  hipMalloc((void**)&DD_dev,NB_HISTO_1D*sizeof(unsigned long long));
  hipMemcpy(DD_dev,DD,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyHostToDevice);
  hipMalloc((void**)&DR_dev,NB_HISTO_1D*sizeof(unsigned long long));
  hipMemcpy(DR_dev,DR,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyHostToDevice);
  hipMalloc((void**)&RR_dev,NB_HISTO_1D*sizeof(unsigned long long));
  hipMemcpy(RR_dev,RR,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyHostToDevice);

  printf("  Correlating \n");
  hipEventRecord(start,0); //Time 0
  cudaCrossAngPM<<<n_blocks,NB_HISTO_1D>>>(npix,pix_full_dev,
					   pos_dev,npD_dev,npR_dev,
					   DD_dev,DR_dev,RR_dev);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elaptime,start,stop);
  printf("  CUDA: Time ellapsed: %3.1f ms\n",elaptime); //Time 1

  //Copy histogram back to host
  hipMemcpy(DD,DD_dev,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  hipMemcpy(DR,DR_dev,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  hipMemcpy(RR,RR_dev,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  //Clean up GPU memory
  hipFree(pos_dev);
  hipFree(npD_dev);
  hipFree(npR_dev);
  hipFree(pix_full_dev);
  hipFree(DD_dev);
  hipFree(DR_dev);
  hipFree(RR_dev);

  //Correct for self-correlations and duplicate pairs
  for(ii=0;ii<NB_HISTO_1D;ii++) {
    DD[ii]/=2;
    RR[ii]/=2;
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

void corr_CUDA_Ang(float cth_min,float cth_max,
		   int npD,int *box_npD,
		   int *box_indD,float *box_posD,
		   int npR,int *box_npR,
		   int *box_indR,float *box_posR,
		   unsigned long long *DD,
		   unsigned long long *DR,
		   unsigned long long *RR)
{
  //////
  // Auto-correlator for angular 2PCF with brute-force
  int *box_npD_dev,*box_npR_dev;
  int *box_indD_dev,*box_indR_dev;
  float *box_posD_dev,*box_posR_dev;
  unsigned long long *DD_dev;
  unsigned long long *DR_dev;
  unsigned long long *RR_dev;
  int ii;

  hipEvent_t start, stop;
  float elaptime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for(ii=0;ii<NB_HISTO_1D;ii++) {
    DD[ii]=0;
    DR[ii]=0;
    RR[ii]=0;
  }

  float thmax=1/I_THETA_MAX;
  hipMemcpyToSymbol(HIP_SYMBOL(cst_nside_cth),&(n_side_cth),sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_nside_phi),&(n_side_phi),sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_cth_min),&(cth_min),sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_cth_max),&(cth_max),sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_thmax),&(thmax),sizeof(float));

  //Allocate GPU memory and copy particle positions
  hipMalloc((void**)&box_posD_dev,3*npD*sizeof(float));
  hipMemcpy(box_posD_dev,box_posD,3*npD*sizeof(float),hipMemcpyHostToDevice);
  hipMalloc((void**)&box_posR_dev,3*npR*sizeof(float));
  hipMemcpy(box_posR_dev,box_posR,3*npR*sizeof(float),hipMemcpyHostToDevice);

  //Allocate and copy box #particles
  hipMalloc((void**)&box_npD_dev,n_boxes2D*sizeof(int));
  hipMemcpy(box_npD_dev,box_npD,n_boxes2D*sizeof(int),hipMemcpyHostToDevice);
  hipMalloc((void**)&box_npR_dev,n_boxes2D*sizeof(int));
  hipMemcpy(box_npR_dev,box_npR,n_boxes2D*sizeof(int),hipMemcpyHostToDevice);

  //Allocate and copy box 1st particle indices
  hipMalloc((void**)&box_indD_dev,n_boxes2D*sizeof(int));
  hipMemcpy(box_indD_dev,box_indD,n_boxes2D*sizeof(int),hipMemcpyHostToDevice);
  hipMalloc((void**)&box_indR_dev,n_boxes2D*sizeof(int));
  hipMemcpy(box_indR_dev,box_indR,n_boxes2D*sizeof(int),hipMemcpyHostToDevice);

  //Allocate GPU memory for the GPU histogram
  hipMalloc((void**)&DD_dev,NB_HISTO_1D*sizeof(unsigned long long));
  hipMemcpy(DD_dev,DD,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyHostToDevice);
  hipMalloc((void**)&DR_dev,NB_HISTO_1D*sizeof(unsigned long long));
  hipMemcpy(DR_dev,DR,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyHostToDevice);
  hipMalloc((void**)&RR_dev,NB_HISTO_1D*sizeof(unsigned long long));
  hipMemcpy(RR_dev,RR,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyHostToDevice);

  printf("  Auto-correlating data \n");
  hipEventRecord(start,0); //Time 0
  cudaCrossAng<<<n_blocks,NB_HISTO_1D>>>(npD,box_posD_dev,
					 box_npD_dev,box_indD_dev,box_posD_dev,
					 DD_dev);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elaptime,start,stop);
  printf("  CUDA: Time ellapsed: %3.1f ms\n",elaptime); //Time 1

  printf("  Auto-correlating random \n");
  hipEventRecord(start,0); //Time 0
  cudaCrossAng<<<n_blocks,NB_HISTO_1D>>>(npR,box_posR_dev,
					 box_npR_dev,box_indR_dev,box_posR_dev,
					 RR_dev);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elaptime,start,stop);
  printf("  CUDA: Time ellapsed: %3.1f ms\n",elaptime); //Time 1

  printf("  Cross-correlating \n");
  hipEventRecord(start,0); //Time 0
  cudaCrossAng<<<n_blocks,NB_HISTO_1D>>>(npD,box_posD_dev,
					 box_npR_dev,box_indR_dev,box_posR_dev,
					 DR_dev);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elaptime,start,stop);
  printf("  CUDA: Time ellapsed: %3.1f ms\n",elaptime); //Time 1

  //Copy histogram back to host
  hipMemcpy(DD,DD_dev,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  hipMemcpy(DR,DR_dev,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  hipMemcpy(RR,RR_dev,NB_HISTO_1D*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  //Clean up GPU memory
  hipFree(box_npD_dev);
  hipFree(box_npR_dev);
  hipFree(box_indD_dev);
  hipFree(box_indR_dev);
  hipFree(box_posD_dev);
  hipFree(box_posR_dev);
  hipFree(DD_dev);
  hipFree(DR_dev);
  hipFree(RR_dev);

  //Correct for self-correlations and duplicate pairs
#ifndef _LOGBIN
  DD[0]-=npD;
  RR[0]-=npR;
#endif //_LOGBIN
  for(ii=0;ii<NB_HISTO_1D;ii++) {
    DD[ii]/=2;
    RR[ii]/=2;
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

void corr_CUDA_3D(float *pos_min,
		  int npD,int *box_npD,
		  int *box_indD,float *box_posD,
		  int npR,int *box_npR,
		  int *box_indR,float *box_posR,
		  unsigned long long *DD,
		  unsigned long long *DR,
		  unsigned long long *RR,
		  int ctype)
{
  //////
  // Auto-correlator for angular 2PCF with brute-force
  int nbns;
  int *box_npD_dev,*box_npR_dev;
  int *box_indD_dev,*box_indR_dev;
  float *box_posD_dev,*box_posR_dev;
  unsigned long long *DD_dev;
  unsigned long long *DR_dev;
  unsigned long long *RR_dev;
  int ii;
  double rmax;

  hipEvent_t start, stop;
  float elaptime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  if(ctype==2) {
    nbns=NB_HISTO_1D;
    rmax=1/I_R_MAX;
  }
  else if(ctype==3) {
    nbns=NB_HISTO_2D*NB_HISTO_2D;
    rmax=sqrt(1/(I_RT_MAX*I_RT_MAX)+1/(I_RL_MAX*I_RL_MAX));
  }
  else if(ctype==4) {
    nbns=NB_HISTO_2D*NB_HISTO_2D;
    rmax=1/I_R3D_MAX;
  }
  else {
    fprintf(stderr,"WTF\n");
    exit(1);
  }

  hipMemcpyToSymbol(HIP_SYMBOL(cst_nside_x),&(n_side[0]),sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_nside_y),&(n_side[1]),sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_nside_z),&(n_side[2]),sizeof(int));
  int irange_x=(int)(rmax*n_side[0]/l_box[0])+1;
  int irange_y=(int)(rmax*n_side[1]/l_box[1])+1;
  int irange_z=(int)(rmax*n_side[2]/l_box[2])+1;
  hipMemcpyToSymbol(HIP_SYMBOL(cst_irange_x),&irange_x,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_irange_y),&irange_y,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_irange_z),&irange_z,sizeof(int));
  float lbx=(float)(l_box[0]);
  float lby=(float)(l_box[1]);
  float lbz=(float)(l_box[2]);
  hipMemcpyToSymbol(HIP_SYMBOL(cst_l_box_x),&lbx,sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_l_box_y),&lby,sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_l_box_z),&lbz,sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_x_min),&(pos_min[0]),sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_y_min),&(pos_min[1]),sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cst_z_min),&(pos_min[2]),sizeof(float));

  for(ii=0;ii<nbns;ii++) {
    DD[ii]=0;
    DR[ii]=0;
    RR[ii]=0;
  }

  //Allocate GPU memory and copy particle positions
  hipMalloc((void**)&box_posD_dev,3*npD*sizeof(float));
  hipMemcpy(box_posD_dev,box_posD,3*npD*sizeof(float),hipMemcpyHostToDevice);
  hipMalloc((void**)&box_posR_dev,3*npR*sizeof(float));
  hipMemcpy(box_posR_dev,box_posR,3*npR*sizeof(float),hipMemcpyHostToDevice);

  //Allocate and copy box #particles
  hipMalloc((void**)&box_npD_dev,n_boxes3D*sizeof(int));
  hipMemcpy(box_npD_dev,box_npD,n_boxes3D*sizeof(int),hipMemcpyHostToDevice);
  hipMalloc((void**)&box_npR_dev,n_boxes3D*sizeof(int));
  hipMemcpy(box_npR_dev,box_npR,n_boxes3D*sizeof(int),hipMemcpyHostToDevice);

  //Allocate and copy box 1st particle indices
  hipMalloc((void**)&box_indD_dev,n_boxes3D*sizeof(int));
  hipMemcpy(box_indD_dev,box_indD,n_boxes3D*sizeof(int),hipMemcpyHostToDevice);
  hipMalloc((void**)&box_indR_dev,n_boxes3D*sizeof(int));
  hipMemcpy(box_indR_dev,box_indR,n_boxes3D*sizeof(int),hipMemcpyHostToDevice);

  //Allocate GPU memory for the GPU histogram
  hipMalloc((void**)&DD_dev,nbns*sizeof(unsigned long long));
  hipMemcpy(DD_dev,DD,nbns*sizeof(unsigned long long),hipMemcpyHostToDevice);
  hipMalloc((void**)&DR_dev,nbns*sizeof(unsigned long long));
  hipMemcpy(DR_dev,DR,nbns*sizeof(unsigned long long),hipMemcpyHostToDevice);
  hipMalloc((void**)&RR_dev,nbns*sizeof(unsigned long long));
  hipMemcpy(RR_dev,RR,nbns*sizeof(unsigned long long),hipMemcpyHostToDevice);

  //HERE
  int jj;
  printf("  Auto-correlating data \n");
  hipEventRecord(start,0); //Time 0
  if(ctype==2) {
    cudaCrossMono<<<n_blocks,NB_HISTO_1D>>>(npD,box_posD_dev,
					    box_npD_dev,box_indD_dev,box_posD_dev,
					    DD_dev);
  }
  else {
    for(jj=0;jj<NB_HISTO_2D/NB_X_BATCH;jj++) {
      dim3 thr(NB_HISTO_2D,NTH_RWS_2D);
      if(ctype==3) {
	cudaCross3Dps<<<n_blocks,thr>>>(npD,box_posD_dev,
					box_npD_dev,box_indD_dev,box_posD_dev,
					DD_dev,jj);
      }
      else {
	cudaCross3Drm<<<n_blocks,thr>>>(npD,box_posD_dev,
					box_npD_dev,box_indD_dev,box_posD_dev,
					DD_dev,jj);
      }
    }
  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elaptime,start,stop);
  printf("  CUDA: Time ellapsed: %3.1f ms\n",elaptime); //Time 1

  printf("  Auto-correlating random \n");
  hipEventRecord(start,0); //Time 0
  if(ctype==2) {
    cudaCrossMono<<<n_blocks,NB_HISTO_1D>>>(npR,box_posR_dev,
					    box_npR_dev,box_indR_dev,box_posR_dev,
					    RR_dev);
  }
  else {
    for(jj=0;jj<NB_HISTO_2D/NB_X_BATCH;jj++) {
      dim3 thr(NB_HISTO_2D,NTH_RWS_2D);
      if(ctype==3) {
	cudaCross3Dps<<<n_blocks,thr>>>(npR,box_posR_dev,
					box_npR_dev,box_indR_dev,box_posR_dev,
					RR_dev,jj);
      }
      else {
	cudaCross3Drm<<<n_blocks,thr>>>(npR,box_posR_dev,
					box_npR_dev,box_indR_dev,box_posR_dev,
					RR_dev,jj);
      }
    }
  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elaptime,start,stop);
  printf("  CUDA: Time ellapsed: %3.1f ms\n",elaptime); //Time 1

  printf("  Cross-correlating \n");
  hipEventRecord(start,0); //Time 0
  if(ctype==2) {
    cudaCrossMono<<<n_blocks,NB_HISTO_1D>>>(npD,box_posD_dev,
					    box_npR_dev,box_indR_dev,box_posR_dev,
					    DR_dev);
  }
  else {
    for(jj=0;jj<NB_HISTO_2D/NB_X_BATCH;jj++) {
      dim3 thr(NB_HISTO_2D,NTH_RWS_2D);
      if(ctype==3) {
	cudaCross3Dps<<<n_blocks,thr>>>(npD,box_posD_dev,
					box_npR_dev,box_indR_dev,box_posR_dev,
					DR_dev,jj);
      }
      else {
	cudaCross3Drm<<<n_blocks,thr>>>(npD,box_posD_dev,
					box_npR_dev,box_indR_dev,box_posR_dev,
					DR_dev,jj);
      }
    }
  }
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elaptime,start,stop);
  printf("  CUDA: Time ellapsed: %3.1f ms\n",elaptime); //Time 1

  //Copy histogram back to host
  hipMemcpy(DD,DD_dev,nbns*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  hipMemcpy(DR,DR_dev,nbns*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  hipMemcpy(RR,RR_dev,nbns*sizeof(unsigned long long),hipMemcpyDeviceToHost);
  //Clean up GPU memory
  hipFree(box_npD_dev);
  hipFree(box_npR_dev);
  hipFree(box_indD_dev);
  hipFree(box_indR_dev);
  hipFree(box_posD_dev);
  hipFree(box_posR_dev);
  hipFree(DD_dev);
  hipFree(DR_dev);
  hipFree(RR_dev);

  //Correct for self-correlations and duplicate pairs
  if(ctype==3) {
    DD[0]-=npD;
    RR[0]-=npR;
  }
  else {
#ifndef _LOGBIN
    DD[0]-=npD;
    RR[0]-=npR;
#endif //_LOGBIN
  }
  for(ii=0;ii<nbns;ii++) {
    DD[ii]/=2;
    RR[ii]/=2;
  }
  
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
